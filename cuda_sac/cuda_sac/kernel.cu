#pragma once

#include <iostream>
#include <string>
#include "cudasac.cuh"
#undef DOMDocument

#include "XBuilder.h"

using namespace std;
using namespace cudacp;
const string X_PATH = "BMPath.xml";

int main()
{
	XBuilder path_builder(X_PATH, XRT_BM_PATH);
	string bm_path = path_builder.GetBMFile();
	cout << bm_path << endl;
	XBuilder builder(bm_path, XRT_BM);
	XModel* xmodel = new XModel();
	builder.GenerateModelFromXml(xmodel);
	float build_time = BuidBitModel32bit(xmodel);
	float exe_time = SACGPU();
	DelGPUModel();

	delete xmodel;
	xmodel = NULL;
	printf("Build time = %f\n", build_time);
	printf("Execution time = %f\n", exe_time);
	printf("---end---\n");
	return 0;
}

