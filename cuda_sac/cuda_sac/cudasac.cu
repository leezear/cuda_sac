#pragma once
#include <hip/hip_runtime.h>

//// CUDA Runtime
//#include <cuda_runtime.h>
//#include <device_functions.h>
//#include <device_launch_parameters.h>
//// Utilities and system includes
//#include <helper_cuda.h>
//#include <helper_functions.h>
//#include <iostream>
//#include <math.h>
//#include "XModel.h"
//#include "math_functions.hpp"
//#include "cuda_runtime_api.h"
//
//using namespace cudacp;
//
//#ifndef MIN
//#define MIN(x,y) ((x < y) ? x : y)
//#endif
//
//static const int WORK_SIZE = 64;
//static const int U32_SIZE = sizeof(u32); ///<4
//static const int U32_BIT = U32_SIZE * 8;	///<32
//static const int U32_POS = 5;
//static const int U32_MOD_MASK = 31;
//
///// A single variable domain needs how many int, it can also be used in locate the index by variable's id;
//__constant__ int D_BITDOM_INTSIZE;
//__constant__ int D_NUM_BD_BLOCK;
//__constant__ int D_NUM_CS_SIZE_BLOCKS;
////////////////////////////////////////////////////////////////////////////
////	一些GPU常量
////////////////////////////////////////////////////////////////////////////
//__device__ __managed__ int NUM_BD_BLOCK;
//__device__ __managed__ int NUM_CS_SIZE_BLOCKS;
//// 一个变量论域的int长度
//__device__ __managed__ int BITDOM_INTSIZE;
//// 整个变量集合的论域的int长度 
//__device__ __managed__ int BITDOMS_INTSIZE;
//// 一个约束的bitsup的int长度 
//__device__ __managed__ int BITSUP_INTSIZE;
//// 整个约束集合的bitsup的int长度
//__device__ __managed__ int BITSUPS_INTSIZE;
////子问题论域总长度
//__device__ __managed__ int BITSUBDOMS_INTSIZE;
//
////////////////////////////////////////////////////////////////////////////
// //GPU约束记录信息，不可更改
////////////////////////////////////////////////////////////////////////////
////	每个变量的int大小
//__device__ __managed__ int *vars_size;
//// 存储约束的scope，类型int3，scope.x: x.id; scope.y: y.id; scope.z: c.id
//__device__ __managed__ int3* scope;
//// 最大dom
//__device__ __managed__ int MAX_DOM_SIZE;
//// subCon长度
//__device__ __managed__ int SUBCON_SIZE;
//
//
////__device__ __managed__ int BITDOM_SIZE;
////__device__ __managed__ int
//// 主问题数据结构，使用UM
//// 表示约束网络论域
//__device__ __managed__ u32* bitDom;
//// 表示约束，不可修改
//__device__ __managed__ u32* bitSup;
////类似队列，存储约束id
//__device__ __managed__ int *mainCon;
////子问题数据结构
////表示子问题的约束网络论域。
//__device__ __managed__ u32* bitSubDom;
////类似队列，存储子问题约束id subCon.x: variable，subCon.y: value，subCon.z: c.id
//__device__ __managed__ ushort3* subCon;
////标记需要归约的约束id，初始全部为1
//__device__ __managed__ int* mainEvtCon;
////标记发生改动的变量id，初始化全部为0
//__device__ __managed__ int* mainEvtVar;
//
//__device__ __managed__ ushort4* subVar;
////标记子问题发生改动的变量id，初始化全部为0
//__device__ __managed__ unsigned short* subEvtVar;
////标记子问题发生改动的约束id，初始化全部为1
//__device__ __managed__ int* subEvtCon;
//
//static const u32 U32_MASK1[32] = {
//	0x80000000, 0x40000000, 0x20000000, 0x10000000,
//	0x08000000, 0x04000000, 0x02000000, 0x01000000,
//	0x00800000, 0x00400000, 0x00200000, 0x00100000,
//	0x00080000, 0x00040000, 0x00020000, 0x00010000,
//	0x00008000, 0x00004000, 0x00002000, 0x00001000,
//	0x00000800, 0x00000400, 0x00000200, 0x00000100,
//	0x00000080, 0x00000040, 0x00000020, 0x00000010,
//	0x00000008, 0x00000004, 0x00000002, 0x00000001,
//};
//
//static const u32 U32_MASK0[32] = {
//	0x7FFFFFFF, 0xBFFFFFFF, 0xDFFFFFFF, 0xEFFFFFFF,
//	0xF7FFFFFF, 0xFBFFFFFF, 0xFDFFFFFF, 0xFEFFFFFF,
//	0xFF7FFFFF, 0xFFBFFFFF, 0xFFDFFFFF, 0xFFEFFFFF,
//	0xFFF7FFFF, 0xFFFBFFFF, 0xFFFDFFFF, 0xFFFEFFFF,
//	0xFFFF7FFF, 0xFFFFBFFF, 0xFFFFDFFF, 0xFFFFEFFF,
//	0xFFFFF7FF, 0xFFFFFBFF, 0xFFFFFDFF, 0xFFFFFEFF,
//	0xFFFFFF7F, 0xFFFFFFBF, 0xFFFFFFDF, 0xFFFFFFEF,
//	0xFFFFFFF7, 0xFFFFFFFB, 0xFFFFFFFD, 0xFFFFFFFE,
//};
//
////__forceinline__ int  GetBitDomIndex(int var_id)
////{
////	return var_id * BITDOM_INTSIZE;
////}
//#define GetBitDomIndex(i,j) (i * BITDOM_INTSIZE + j)
//
//__inline__ __device__ __host__ int GetTopNum(int num_elements, int num_threads)
//{
//	return (num_elements + (num_threads - 1)) / num_threads;
//}
////************************************
//// Method:    intsizeof
//// FullName:  intsizeof
//// Access:    public 
//// Returns:   int
//// Qualifier: 获取用bit表示的int长度
//// Parameter: const int x
////************************************
//inline int intsizeof(const int x)
//{
//	return (int)ceil((float)x / U32_BIT);
//}
//
//#define GetBitSupIndexByINTPrstn(cid,x_val,y_val) (cid * BITSUP_INTSIZE + x_val * MAX_DOM_SIZE + y_val)
//#define GetBitSupIndexByTuple(cid,t) (cid * BITSUP_INTSIZE + t.x  + t.y / U32_BIT * MAX_DOM_SIZE)
//void DelGPUModel();
//
//void BuildBitModel(XModel *xm)
//{
//#pragma region 计算常量
//	BITDOM_INTSIZE = intsizeof(xm->feature.max_dom_size);
//	MAX_DOM_SIZE = xm->feature.max_dom_size;
//	BITDOMS_INTSIZE = BITDOM_INTSIZE * xm->feature.vs_size;
//	BITSUP_INTSIZE = xm->feature.max_dom_size * BITDOM_INTSIZE;
//	BITSUPS_INTSIZE = BITSUP_INTSIZE * xm->feature.cs_size;
//	BITSUBDOMS_INTSIZE = xm->feature.vs_size * xm->feature.max_dom_size * BITDOMS_INTSIZE;
//	SUBCON_SIZE = xm->feature.vs_size * xm->feature.ds_size * xm->feature.cs_size;
//#pragma endregion 计算常量
//#pragma region 约束网络信息
//	cudaMallocManaged(&vars_size, sizeof(int) * xm->feature.vs_size);
//	// 初始化变量域大小
//	for (int i = 0; i < xm->feature.vs_size; ++i)
//	{
//		XVar* v = xm->vars[i];
//		XDom* d = xm->doms[v->dom_id];
//		vars_size[i] = d->size;
//	}
//
//	// 初始化scope
//	cudaMallocManaged(&scope, sizeof(int3) * xm->feature.cs_size);
//	for (int i = 0; i < xm->feature.cs_size; ++i)
//	{
//		XCon *c = xm->cons[i];
//		scope[i].x = c->scope[0];
//		scope[i].y = c->scope[1];
//		scope[i].z = c->id;
//	}
//#pragma endregion 约束网络信息
//#pragma region 拷贝bitDom
//	cudaMallocManaged(&bitDom, sizeof(int) * BITDOMS_INTSIZE);
//	cudaMallocManaged(&mainEvtVar, sizeof(int) * xm->feature.vs_size);
//
//	for (int i = 0; i < xm->feature.vs_size; ++i)
//	{
//		XVar* v = xm->vars[i];
//		XDom* d = xm->doms[v->dom_id];
//		int dom_size = d->size;
//		int dom_int_size = intsizeof(dom_size);
//
//		for (int j = 0; j < BITDOM_INTSIZE; ++j)
//		{
//			int idx = GetBitDomIndex(i, j);
//
//			if (j < dom_int_size - 1)
//			{
//				bitDom[idx] = UINT32_MAX;
//			}
//			else if (j == dom_int_size - 1)
//			{
//				int offset = U32_BIT - (dom_size & U32_MOD_MASK);
//				bitDom[idx] = UINT32_MAX << offset;
//			}
//			else
//			{
//				bitDom[idx] = 0;
//			}
//		}
//
//		mainEvtVar[i] = 1;
//	}
//
//	//for (int i = 0; i < xm->feature.vs_size; ++i)
//	//{
//	//	for (int j = 0; j < BITDOM_INTSIZE; ++j)
//	//	{
//	//		int idx = GetBitDomIndex(i, j);
//	//		printf("var = %d, j = %d, idx = %d, pre= %x\n", i, j, idx, bitDom[idx]);
//	//	}
//	//}
//#pragma endregion 拷贝bitDom
//#pragma region 创建bitSubDom
//
//#pragma endregion 创建bitSubDom
//#pragma region 拷贝bitCon
//	cudaMallocManaged(&bitSup, sizeof(int) * BITSUPS_INTSIZE);
//	for (int i = 0; i < xm->feature.cs_size; ++i)
//	{
//		XCon* c = xm->cons[i];
//		XRel* r = xm->rels[c->rel_id];
//		XVar* v[2] = { xm->vars[c->scope[0]],xm->vars[c->scope[1]] };
//		XDom* d[2] = { xm->doms[v[0]->dom_id], xm->doms[v[1]->dom_id] };
//
//		// 初始化位矩阵
//		for (int j = 0; j < BITDOM_INTSIZE; ++j)
//		{
//			for (int k = 0; k < xm->feature.max_dom_size; ++k)
//			{
//				const int idx = GetBitSupIndexByINTPrstn(c->id, j, k);
//				if (k < d[0]->size && (j < (d[1]->size / U32_BIT)))
//				{
//					// 支持取0x0000...,冲突取0xFFF...
//					bitSup[idx] = (r->sem == SEM_CONFLICT) ? UINT32_MAX : 0;
//				}
//				else if (j == (d[1]->size / U32_BIT))
//				{
//					bitSup[idx] = (r->sem == SEM_CONFLICT) ? UINT32_MAX : 0;
//					bitSup[idx] <<= U32_BIT - (d[1]->size & U32_MOD_MASK);
//				}
//				else
//				{
//					bitSup[idx] = 0;
//				}
//			}
//		}
//
//		// 向位矩阵中填充值
//		for (int j = 0; j < r->size; ++j)
//		{
//			int2 t;
//			t.x = r->tuples[j][0];
//			t.y = r->tuples[j][1];
//
//			const int idx = GetBitSupIndexByTuple(c->id, t);
//
//			if (r->sem == SEM_SUPPORT)
//			{
//				bitSup[idx] |= U32_MASK1[t.y & U32_MOD_MASK];
//			}
//			else
//			{
//				bitSup[idx] &= U32_MASK0[t.y & U32_MOD_MASK];
//			}
//		}
//		//// 初始化位矩阵
//		//for (int j = 0; j < BITDOM_INTSIZE; ++j)
//		//{
//		//	for (int k = 0; k < xm->feature.max_dom_size; ++k)
//		//	{
//		//		const int idx = GetBitSupIndexByINTPrstn(c->id, j, k);
//		//		printf("c_id = %d, j = %d, k = %d, idx = %d, pre= %x\n", i, j, k, idx, bitSup[idx]);
//		//	}
//		//}
//	}
//#pragma endregion 拷贝bitCon
//#pragma region mainCon
//
//	cudaMallocManaged(&mainCon, sizeof(int)*xm->feature.cs_size);
//	cudaMallocManaged(&mainEvtCon, sizeof(int) * xm->feature.cs_size);
//
//	for (int i = 0; i < xm->feature.cs_size; ++i)
//	{
//		mainCon[i] = i;
//		mainEvtCon[i] = 0;
//	}
//
//#pragma endregion mainCon
//#pragma region subCon
//	cudaMallocManaged(&subCon, sizeof(int3)*SUBCON_SIZE);
//	for (int i = 0; i < xm->feature.vs_size; ++i)
//	{
//		XVar* v = xm->vars[i];
//		for (int j = 0; j < xm->feature.max_dom_size; ++j)
//		{
//			for (int k = 0; k < xm->feature.cs_size; ++k)
//			{
//				const int idx = i*xm->feature.max_dom_size*xm->feature.cs_size + j*xm->feature.cs_size + k;
//
//				subCon[idx].x = i;
//				subCon[idx].y = j;
//				subCon[idx].z = k;
//			}
//		}
//	}
//#pragma endregion subCon
//#pragma region 创建子问题数据类型
//#pragma endregion 创建子问题数据类型
//}
//
//void DelGPUModel()
//{
//	cudaFree(vars_size);
//	cudaFree(scope);
//	cudaFree(bitDom);
//	cudaFree(bitSup);
//}



