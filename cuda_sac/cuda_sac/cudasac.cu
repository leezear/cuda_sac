
#include <hip/hip_runtime.h>
////#pragma once
//// CUDA Runtime
////#include <cuda_runtime.h>
////#include <device_functions.h>
////#include <device_launch_parameters.h>
//// Utilities and system includes
////#include <helper_cuda.h>
////#include <helper_functions.h>
////#include <thrust/device_vector.h>
////#include <thrust/scan.h>
////#include <iostream>
////#include <math.h>
////#include "XModel.h"
////#include "math_functions.hpp"
////#include "cuda_runtime_api.h"
////
////using namespace cudacp;
////
////#ifndef MIN
////#define MIN(x,y) ((x < y) ? x : y)
////#endif
////// num_threads
////static const int num_threads = 128;
////static const int U32_SIZE = sizeof(u32); ///<4
////static const int U32_BIT = U32_SIZE * 8;	///<32
////static const int U32_POS = 5;
////static const int U32_MOD_MASK = 31;
////
////struct int_predicate
////{
////	__host__ __device__	bool operator()(const int x)
////	{
////		return x > 0;
////	}
////};
////
////// һ��bitDom[x]�ĳ���
////__constant__ int D_BITDOM_INTSIZE;
////// ����bitDom�ĳ���
////
////__constant__ int D_BITDOMS_INTSIZE;
////
////// bit֧�֣� uint2 bitSup[c][a][idx].x = bitSup[c,x,a,idx]
////// bit֧�֣� uint2 bitSup[c][a][idx].y = bitSup[c,y,a,idx]
//////__device__ uint2*** d_bitSup;
//////__host__ uint2*** h_bitSup;
//////__device__ u32** d_bitDom;
//////__host__ u32** h_bitDom;
//////__device__ u32**
////__constant__ int D_NUM_BD_BLOCK;
////__constant__ int D_NUM_CS_SIZE_BLOCKS;
//////////////////////////////////////////////////////////////////////////////
//////	һЩGPU����
//////////////////////////////////////////////////////////////////////////////
////__device__ __managed__ int NUM_BD_BLOCK;
////__device__ __managed__ int NUM_CS_SIZE_BLOCKS;
////// һ�����������int����
////__device__ __managed__ int BITDOM_INTSIZE;
////// �����������ϵ������int���� 
////__device__ __managed__ int BITDOMS_INTSIZE;
////// һ��Լ����bitsup��int���� 
////__device__ __managed__ int BITSUP_INTSIZE;
////// ����Լ�����ϵ�bitsup��int����
////__device__ __managed__ int BITSUPS_INTSIZE;
//////�����������ܳ���
////__device__ __managed__ int BITSUBDOMS_INTSIZE;
//////��������
////__device__ __managed__ int VS_SIZE;
//////Լ������
////__device__ __managed__ int CS_SIZE;
//////������Լ��ѹ��BLOCK��
////__device__ __managed__ int MCC_BLOCK;
//////////////////////////////////////////////////////////////////////////////
//////	һЩGPU����
//////////////////////////////////////////////////////////////////////////////
////__device__ __managed__ int M_Qsize;
////
//////////////////////////////////////////////////////////////////////////////
//////  GPUԼ����¼��Ϣ�����ɸ���
//////////////////////////////////////////////////////////////////////////////
//////	ÿ��������int��С
////__device__ __managed__ int *vars_size;
////// �洢Լ����scope������int3��scope.x: x.id; scope.y: y.id; scope.z: c.id
////__device__ __managed__ int3* scope;
////// ���dom
////__device__ __managed__ int MAX_DOM_SIZE;
////// subCon����
////__device__ __managed__ int SUBCON_SIZE;
////
////
//////__device__ __managed__ int BITDOM_SIZE;
//////__device__ __managed__ int
////// ���������ݽṹ��ʹ��UM
////// ��ʾԼ����������
////__device__ __managed__ u32* bitDom;
////// ��ʾԼ���������޸�
////__device__ __managed__ uint2* bitSup;
////////���ƶ��У��洢Լ��id
//////__device__ __managed__ int *mainCon;
////////���������ݽṹ
//////��ʾ�������Լ����������
////__device__ __managed__ u32* bitSubDom;
////////���ƶ��У��洢������Լ��id subCon.x: variable��subCon.y: value��subCon.z: c.id
//////__device__ __managed__ ushort3* subCon;
//////���������������Ƿ�ɾ������ʼ��ȫ��Ϊ1
////__device__ __managed__ int* M_VarPre;
//////���������Լ���Ƿ����飬��ʼ��ȫ��Ϊ1
////__device__ __managed__ int* M_ConPre;
//////������Լ����������(ѹ����)
////__device__ __managed__ int* M_ConEvt;
//////������Լ����������
////__device__ __managed__ int* M_Con;
////
//////���������������Ƿ�ɾ������ʼ��ȫ��Ϊ1
////__device__ __managed__ int* S_VarPre;
//////��¼�����������
////__device__ __managed__ uint3* S_Var;
//////���������Լ���Ƿ����飬��ʼ��ȫ��Ϊ1
////__device__ __managed__ int* S_ConPre;
//////������Լ����������(ѹ����)
////__device__ __managed__ int3* S_ConEvt;
//////������Լ����������
////__device__ __managed__ int3* S_Con;
//////
////int* MCC_BlocksCount;
////int* MCC_BlocksOffset;
////
////thrust::device_vector<int> MCC_BCount;
////thrust::device_vector<int> MCC_BOffset;
//////
//////__device__ __managed__ ushort4* subVar;
////////��������ⷢ���Ķ��ı���id����ʼ��ȫ��Ϊ0
//////__device__ __managed__ unsigned short* subEvtVar;
////////��������ⷢ���Ķ���Լ��id����ʼ��ȫ��Ϊ1
//////__device__ __managed__ int* subEvtCon;
////
////static const u32 U32_MASK1[32] = {
////	0x80000000, 0x40000000, 0x20000000, 0x10000000,
////	0x08000000, 0x04000000, 0x02000000, 0x01000000,
////	0x00800000, 0x00400000, 0x00200000, 0x00100000,
////	0x00080000, 0x00040000, 0x00020000, 0x00010000,
////	0x00008000, 0x00004000, 0x00002000, 0x00001000,
////	0x00000800, 0x00000400, 0x00000200, 0x00000100,
////	0x00000080, 0x00000040, 0x00000020, 0x00000010,
////	0x00000008, 0x00000004, 0x00000002, 0x00000001,
////};
////
////static const u32 U32_MASK0[32] = {
////	0x7FFFFFFF, 0xBFFFFFFF, 0xDFFFFFFF, 0xEFFFFFFF,
////	0xF7FFFFFF, 0xFBFFFFFF, 0xFDFFFFFF, 0xFEFFFFFF,
////	0xFF7FFFFF, 0xFFBFFFFF, 0xFFDFFFFF, 0xFFEFFFFF,
////	0xFFF7FFFF, 0xFFFBFFFF, 0xFFFDFFFF, 0xFFFEFFFF,
////	0xFFFF7FFF, 0xFFFFBFFF, 0xFFFFDFFF, 0xFFFFEFFF,
////	0xFFFFF7FF, 0xFFFFFBFF, 0xFFFFFDFF, 0xFFFFFEFF,
////	0xFFFFFF7F, 0xFFFFFFBF, 0xFFFFFFDF, 0xFFFFFFEF,
////	0xFFFFFFF7, 0xFFFFFFFB, 0xFFFFFFFD, 0xFFFFFFFE,
////};
////
//////__forceinline__ int  GetBitDomIndex(int var_id)
//////{
//////	return var_id * BITDOM_INTSIZE;
//////}
////
////// ����x��index���bitDomλ��
////#define GetBitDomIndex(x, i) (x * BITDOM_INTSIZE + i)
////// ��������������ֵ�ֵ�ĸ�����ȡbit��ʾ��ƫ����
////#define GetOffSet(x)(U32_BIT - (x & U32_MOD_MASK))
////
////#define GetBitSubDomStartIndex(x,a)((x * MAX_DOM_SIZE + a) * BITDOMS_INTSIZE)
////#define GetBitSubDomIndex(x, a, y, i)(GetBitSubDomStartIndex(x,a) + GetBitDomIndex(y, i))
////
////__device__ bool IsGtZero(int x)
////{
////	return x > 0;
////}
////
////__inline__ __device__ __host__ int GetTopNum(int num_elements, int num_threads)
////{
////	return (num_elements + (num_threads - 1)) / num_threads;
////}
//////************************************
////// Method:    intsizeof
////// FullName:  intsizeof
////// Access:    public 
////// Returns:   int
////// Qualifier: ��ȡ��bit��ʾ��int����
////// Parameter: const int x
//////************************************
////inline int intsizeof(const int x)
////{
////	return (int)ceil((float)x / U32_BIT);
////}
////
////__device__ __inline__ int pow2i(int e)
////{
////	return 1 << e;
////}
//////__global__ void enforceACMain(u32* bitDom, u32* bitSup, u32* M_Con, u32* M_ConEvt, u32* M_ConPre, u32* M_VarPre)
//////{
//////
//////}
////
//////ͨ���Ѹı��������Լ������
////__global__ void GenConPre(int *VarPre, int* BlocksCount, int3* scp, int len)
////{
////	const int idx = blockDim.x*blockIdx.x + threadIdx.x;
////	if (idx < len)
////	{
////		int3 sp = scp[idx];
////		int pred;
////		if (VarPre[sp.x] == 1 || VarPre[sp.y] == 1)
////			pred = 1;
////		else
////			pred = 0;
////
////		int BC = __syncthreads_count(pred);
////
////		if (threadIdx.x == 0)
////		{
////			BlocksCount[threadIdx.x] = BC;
////		}
////	}
////}
////
////__global__ void CompactQ(int *VarPre, int* ConEvt, int* BOffset, int3* scp, int len)
////{
////	int idx = threadIdx.x + blockIdx.x*blockDim.x;
////	extern __shared__ int warpTotals[];
////	һ���߳̿�����128���߳�
////	һ��������4���߳���
////	if (idx < len)
////	{
////		int3 sp = scp[idx];
////		int pred;
////		����ж�
////		if (VarPre[sp.x] == 1 || VarPre[sp.y] == 1)
////			pred = 1;
////		else
////			pred = 0;
////
////		warp index
////		�߳�������
////		int w_i = threadIdx.x / warpSize;
////		thread index within a warp
////		�߳������߳�����
////		int w_l = idx % warpSize;
////		thread mask (ERROR IN THE PAPERminus one is required)
////		�߳�����
////		INT_MAX = 1111 1111 1111 1111 1111 1111 1111 1111 
////		���߳���id=0������32-0-1 = 31λ �Ҳ�ʣ��1λ
////		���߳���id=5������32-5-1 = 26λ �Ҳ�ʣ��6λ
////		���߳���id=31������32-31-1 = 0λ �Ҳ�ʣ��32λ
////		�߳�����threid|  31~~~~~~0
////		ballot��Ӧλ��|   1......1
////		int t_m = INT_MAX >> (warpSize - w_l - 1);
////		balres = number whose ith bit is one if the ith's thread pred is true masked up to the current index in warp
////		�߳��ھֲ�����pred = 1�������밴λ�뵫���˵��������߳�id�ļ�¼��ֻ��������ǰ����ж�
////		int b = __ballot(pred) & t_m;
////		popc count the number of bit one. simply count the number predicated true BEFORE MY INDEX
////		����ֻ���㵱ǰ�߳�������Ӧ��ǰN����λ��֮��
////		��Ϊ�߳���������ɨ��
////		int t_u = __popc(b);
////
////		��ÿ���߳������һ���߳�д�빲���ڴ棬��ӦidΪ�߳���ID�������߳�ID�ӻأ�
////		��������͵�����ֵд�빲���ڴ棬��������͵�ֵû�б�����
////		warpTotals����Ϊ4
////		if (w_l == warpSize - 1)
////			warpTotals[w_i] = t_u + pred;
////
////		__syncthreads();
////
////		�߳���idΪ0���߳������߳�id����blockDim.x = 128����w_l < 128/32 = 4
////		�߳̿��ڵ�һ���߳�����ǰ��4�����߳���������w_l < ��߳�������4������ÿ���߳�����һ���߳�����
////		if (w_i == 0 && w_l < blockDim.x / warpSize)
////		{
////			int w_i_u = 0;
////			for (int j = 0; j <= 5; ++j)
////			{
////				# of the ones in the j'th digit of the warp offsets
////				0->5 6��λ�ã�
////				000 001
////				000 010
////				000 100
////				001 000
////				010 000
////				100 000
////				int b_j = __ballot(warpTotals[w_l] & pow2i(j));
////				w_i_u += (__popc(b_j & t_m)) << j;
////				printf("indice %i t_m=%i,j=%i,b_j=%i,w_i_u=%i\n",w_l,t_m,j,b_j,w_i_u);
////			}
////			warpTotals[w_l] = w_i_u;
////		}
////		__syncthreads();
////
////		if (pred)
////			ConEvt[t_u + warpTotals[w_i] + BOffset[blockIdx.x]] = scp[idx].z;
////
////	}
////}
////
////void CompactQueueMain()
////{
////	//��Լ����������
////	//P1
////	GenConPre << <MCC_BLOCK, num_threads >> >(M_VarPre, MCC_BlocksCount, scope, CS_SIZE);
////	cudaDeviceSynchronize();
////	//P2
////	thrust::exclusive_scan(MCC_BCount.begin(), MCC_BCount.end(), MCC_BOffset.begin());
////	cudaDeviceSynchronize();
////	//P3
////	//ÿ��Լ��һ���߳̽��й�Լ,�����ڴ��С = һ�������߳����ĸ���,����װ���߳���������
////	CompactQ << <MCC_BLOCK, num_threads, sizeof(int)*(num_threads / warpSize) >> >(M_VarPre, M_Con, MCC_BlocksOffset, scope, CS_SIZE);
////}
////
////#define GetBitSupIndexByINTPrstn(cid,x_val,y_val) (cid * BITSUP_INTSIZE + x_val * BITDOM_INTSIZE + y_val)
////
////__inline__ __device__ __host__ int2 GetBitSupIndexByTuple(int cid, int2 t)
////{
////	return make_int2(cid * BITSUP_INTSIZE + t.x * BITDOM_INTSIZE + (t.y >> U32_POS), cid * BITSUP_INTSIZE + t.y * BITDOM_INTSIZE + (t.x >> U32_POS));
////}
////
////__inline__ __device__ __host__ int GetBitSupIndexById(int cid)
////{
////	return cid * BITSUP_INTSIZE;
////}
////
////void DelGPUModel();
////
////void BuildBitModel(XModel *xm)
////{
////#pragma region ���㳣��
////	BITDOM_INTSIZE = intsizeof(xm->feature.max_dom_size);
////	MAX_DOM_SIZE = xm->feature.max_dom_size;
////	VS_SIZE = xm->feature.vs_size;
////	CS_SIZE = xm->feature.cs_size;
////	BITDOMS_INTSIZE = BITDOM_INTSIZE * VS_SIZE;
////	BITSUP_INTSIZE = MAX_DOM_SIZE * BITDOM_INTSIZE;
////	BITSUPS_INTSIZE = BITSUP_INTSIZE * CS_SIZE;
////	BITSUBDOMS_INTSIZE = VS_SIZE * MAX_DOM_SIZE * BITDOMS_INTSIZE;
////	SUBCON_SIZE = VS_SIZE * MAX_DOM_SIZE * CS_SIZE;
////#pragma endregion ���㳣��
////#pragma region Լ��������Ϣ
////	//cudaMallocManaged(&vars_size, sizeof(int) * VS_SIZE);
////	//// ��ʼ���������С
////	//for (int i = 0; i < xm->feature.vs_size; ++i)
////	//{
////	//	XVar* v = xm->vars[i];
////	//	XDom* d = xm->doms[v->dom_id];
////	//	vars_size[i] = d->size;
////	//}
////
////	// ��ʼ��scope
////	cudaMallocManaged(&scope, sizeof(int3) * CS_SIZE);
////	for (int i = 0; i < CS_SIZE; ++i)
////	{
////		XCon *c = xm->cons[i];
////		scope[i].x = c->scope[0];
////		scope[i].y = c->scope[1];
////		scope[i].z = c->id;
////	}
////
////	////��ʾ
////	//for (int i = 0; i < CS_SIZE; ++i)
////	//{
////	//	printf("scope[%d] = {%d, %d}\n", scope[i].z, scope[i].x, scope[i].y);
////	//}
////#pragma endregion Լ��������Ϣ
////#pragma region ����bitDom
////	cudaMallocManaged(&bitDom, sizeof(u32) * BITDOMS_INTSIZE);
////	cudaMallocManaged(&M_VarPre, sizeof(int) * VS_SIZE);
////
////	for (int i = 0; i < VS_SIZE; ++i)
////	{
////		XVar* v = xm->vars[i];
////		XDom* d = xm->doms[v->dom_id];
////		const int dom_size = d->size;
////		// ��ǰ������ʵ��INT����
////		const int dom_int_size = intsizeof(dom_size);
////
////		for (int j = 0; j < BITDOM_INTSIZE; ++j)
////		{
////			const int idx = GetBitDomIndex(i, j);
////			//printf("idx = %d\n", idx);
////			// �������
////			if (j < dom_int_size - 1)
////				bitDom[idx] = UINT32_MAX;
////			else if (j == dom_int_size - 1)
////				bitDom[idx] = UINT32_MAX << GetOffSet(dom_size);
////			else
////				bitDom[idx] = 0;
////		}
////
////		M_VarPre[i] = 1;
////	}
////
////	//for (int i = 0; i < VS_SIZE; ++i)
////	//{
////	//	for (int j = 0; j < BITDOM_INTSIZE; ++j)
////	//	{
////	//		int idx = GetBitDomIndex(i, j);
////	//		printf("var = %d, j = %d, idx = %d, bitDom = %x, pre= %x\n", i, j, idx, bitDom[idx], M_VarPre[i]);
////	//	}
////	//}
////#pragma endregion ����bitDom
////#pragma region ����bitSubDom
////	cudaMallocManaged(&bitSubDom, sizeof(u32)*BITDOMS_INTSIZE*VS_SIZE*MAX_DOM_SIZE);
////	for (int i = 0; i < VS_SIZE; ++i)
////	{
////		for (int j = 0; j < MAX_DOM_SIZE; ++j)
////		{
////			const int start_idx = GetBitSubDomStartIndex(i, j);
////			for (int k = 0; k < BITSUBDOMS_INTSIZE; ++k)
////				bitSubDom[start_idx + k] = bitDom[k];
////			//���(i,j)��bitDom �ĵ�
////			//��ȡi,j,i����ʼ��ַ��
////			const int ijistart = start_idx + i*BITDOM_INTSIZE;
////			for (int k = 0; k < BITDOM_INTSIZE; ++k)
////				// j������K�ķ�Χ��:j/32,����j%32λ��Ϊ1
////				if (k == j >> U32_POS)
////					bitSubDom[ijistart + k] = U32_MASK1[j&U32_MOD_MASK];
////			//����λ��Ϊ0
////				else
////					bitSubDom[ijistart + k] = 0;
////		}
////	}
////
////	//for (int i = 0; i < VS_SIZE; ++i)
////	//{
////	//	for (int j = 0; j < MAX_DOM_SIZE; ++j)
////	//	{
////	//		printf("sub problem:(%d, %d): ", i, j);
////	//		const int start_idx = GetBitSubDomStartIndex(i, j);
////	//		for (int k = 0; k < BITDOMS_INTSIZE; ++k)
////	//		{
////	//			printf("%x ", bitSubDom[start_idx + k]);
////	//		}
////	//		printf("\n");
////	//	}
////	//}
////#pragma endregion ����bitSubDom
////#pragma region ����bitSup
////	cudaMallocManaged(&bitSup, sizeof(uint2) * BITSUPS_INTSIZE);
////	for (int i = 0; i < CS_SIZE; ++i)
////	{
////		XCon* c = xm->cons[i];
////		XRel* r = xm->rels[c->rel_id];
////		XVar* v[2] = { xm->vars[c->scope[0]], xm->vars[c->scope[1]] };
////		XDom* d[2] = { xm->doms[v[0]->dom_id], xm->doms[v[1]->dom_id] };
////
////		//��ʼ��λ����
////		for (int j = 0; j < MAX_DOM_SIZE; ++j)
////		{
////			for (int k = 0; k < BITDOM_INTSIZE; ++k)
////			{
////				const int idx = GetBitSupIndexByINTPrstn(c->id, j, k);
////				if (j < d[0]->size && (k < (d[1]->size >> U32_POS)))
////				{
////					//֧��ȡ0x0000..., ��ͻȡ0xFFF...
////					bitSup[idx].x = (r->sem == SEM_CONFLICT) ? UINT32_MAX : 0;
////					bitSup[idx].y = (r->sem == SEM_CONFLICT) ? UINT32_MAX : 0;
////				}
////				else if (k == (d[1]->size >> U32_POS))
////				{
////					bitSup[idx].x = (r->sem == SEM_CONFLICT) ? UINT32_MAX : 0;
////					bitSup[idx].y = (r->sem == SEM_CONFLICT) ? UINT32_MAX : 0;
////					bitSup[idx].x <<= U32_BIT - (d[1]->size & U32_MOD_MASK);
////					bitSup[idx].y <<= U32_BIT - (d[1]->size & U32_MOD_MASK);
////				}
////				else
////				{
////					bitSup[idx].x = 0;
////					bitSup[idx].y = 0;
////				}
////			}
////		}
////		//��λ���������ֵ
////		for (int j = 0; j < r->size; ++j)
////		{
////			const int2 t = make_int2(r->tuples[j][0], r->tuples[j][1]);
////			//printf("c_id= %d, %d, %d\n", c->id, t.x, t.y);
////			const int2 idx = GetBitSupIndexByTuple(c->id, t);
////			//printf("idx = %d, %d\n", idx.x, idx.y);
////			if (r->sem == SEM_SUPPORT)
////			{
////				bitSup[idx.x].x |= U32_MASK1[t.y & U32_MOD_MASK];
////				bitSup[idx.y].y |= U32_MASK1[t.x & U32_MOD_MASK];
////			}
////			else
////			{
////				bitSup[idx.x].x &= U32_MASK0[t.y & U32_MOD_MASK];
////				bitSup[idx.y].y &= U32_MASK0[t.x & U32_MOD_MASK];
////			}
////		}
////		//// ��ʼ��λ����
////		//for (int j = 0; j < MAX_DOM_SIZE; ++j)
////		//{
////		//	printf("c_id = %d, j = %d: ", i, j);
////		//	for (int k = 0; k < BITDOM_INTSIZE; ++k)
////		//	{
////		//		const int idx = GetBitSupIndexByINTPrstn(c->id, j, k);
////		//		printf("%x, %x", bitSup[idx].x, bitSup[idx].y);
////		//	}
////		//	printf("\n");
////		//}
////	}
////#pragma endregion ����bitSup
////#pragma region ����Լ��
////	cudaMallocManaged(&M_Con, sizeof(int)*CS_SIZE);
////	cudaMallocManaged(&M_ConEvt, sizeof(int) * CS_SIZE);
////	cudaMallocManaged(&M_ConPre, sizeof(int)*CS_SIZE);
////
////	for (int i = 0; i < CS_SIZE; ++i)
////	{
////		M_Con[i] = i;
////		M_ConEvt[i] = i;
////		M_ConPre[i] = 1;
////		//printf("i = %d , M_Con = %d, M_ConEvt = %d, M_ConPre = %d\n", i, M_Con[i], M_ConEvt[i], M_ConPre[i]);
////	}
////#pragma endregion ����Լ��
////#pragma region ������Լ������
////	cudaMallocManaged(&S_ConPre, sizeof(int)*SUBCON_SIZE);
////	cudaMallocManaged(&S_ConEvt, sizeof(int3)*SUBCON_SIZE);
////	cudaMallocManaged(&S_Con, sizeof(int3)*SUBCON_SIZE);
////	cudaMallocManaged(&S_VarPre, sizeof(int)*VS_SIZE*MAX_DOM_SIZE*VS_SIZE);
////	cudaMallocManaged(&S_Var, sizeof(int3)*VS_SIZE*MAX_DOM_SIZE*VS_SIZE);
////
////	for (int i = 0; i < VS_SIZE; ++i)
////	{
////		const XVar* v = xm->vars[i];
////		for (int j = 0; j < MAX_DOM_SIZE; ++j)
////		{
////			for (int k = 0; k < CS_SIZE; ++k)
////			{
////				// ������(i, j) kΪԼ��id
////				const int idx = (i*MAX_DOM_SIZE + j)*CS_SIZE + k;
////				//i*xm->feature.max_dom_size*xm->feature.cs_size + j*xm->feature.cs_size + k;
////
////				S_Con[idx].x = i;
////				S_Con[idx].y = j;
////				S_Con[idx].z = k;
////
////				S_ConEvt[idx].x = i;
////				S_ConEvt[idx].y = j;
////				S_ConEvt[idx].z = k;
////
////				S_ConPre[idx] = 1;
////				//printf("S_Con = (%d, %d, %d), S_ConEvt = (%d, %d, %d), pre = %d\n", S_Con[idx].x, S_Con[idx].y, S_Con[idx].z, S_ConEvt[idx].x, S_ConEvt[idx].y, S_ConEvt[idx].z, S_ConPre[idx]);
////			}
////
////			for (int k = 0; k < VS_SIZE; ++k)
////			{
////				//������(i, j) kΪ����id
////				const int idx = (i*MAX_DOM_SIZE + j)*VS_SIZE + k;
////				S_Var[idx].x = i;
////				S_Var[idx].y = j;
////				S_Var[idx].z = k;
////
////				S_VarPre[idx] = 1;
////
////				//printf("S_Var = (%d, %d, %d), S_VarPre = %d\n", S_Var[idx].x, S_Var[idx].y, S_Var[idx].z, S_VarPre[idx]);
////			}
////		}
////	}
////#pragma endregion ������Լ������
////
////#pragma region �������й��
////	//���������ѹ����BLOCK��
////	MCC_BLOCK = GetTopNum(CS_SIZE, num_threads);
////	MCC_BCount.resize(MCC_BLOCK, 0);
////	MCC_BOffset.resize(MCC_BLOCK, 0);
////	MCC_BlocksCount = thrust::raw_pointer_cast(MCC_BCount.data());
////	MCC_BlocksOffset = thrust::raw_pointer_cast(MCC_BOffset.data());
////#pragma endregion
////
////}
////
////__global__ void ConCheckMain(int* ConEvt, int* btSp, int2* scp)
////{
////	const int c_id = blockIdx.x;
////	//��ȡԼ����bitSup�Ŀ�ʼ����
////	const int start_idx = GetBitSupIndexById(c_id);
////	const int2 sp = scp[c_id];
////	extern __shared__ int2[];
////}
////
////void ConstraintCheckMain()
////{
////	//num_threads��ÿɱ�
////	ConCheckMain << <CS_SIZE, num_threads >> >();
////}
////
////float SACGPU()
////{
////	//1. ����������ִ��AC
////	//1.1. ��ѹ��
////	CompactQueueMain();
////	//1.2. Լ�����
////	ConstraintCheckMain();
////}
////
////void DelGPUModel()
////{
////	cudaFree(scope);
////	cudaFree(bitDom);
////	cudaFree(M_VarPre);
////	cudaFree(bitSubDom);
////	cudaFree(bitSup);
////	cudaFree(M_Con);
////	cudaFree(M_ConEvt);
////	cudaFree(M_ConPre);
////	cudaFree(S_ConPre);
////	cudaFree(S_ConEvt);
////	cudaFree(S_Con);
////	cudaFree(S_Var);
////	cudaFree(S_VarPre);
////}
////
////
