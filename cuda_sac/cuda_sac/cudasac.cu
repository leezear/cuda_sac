#pragma once
#include <hip/hip_runtime.h>

//// CUDA Runtime
//#include <cuda_runtime.h>
//#include <device_functions.h>
//#include <device_launch_parameters.h>
//// Utilities and system includes
//#include <helper_cuda.h>
//#include <helper_functions.h>
//#include <iostream>
//#include <math.h>
//#include "XModel.h"
//#include "math_functions.hpp"
//#include "cuda_runtime_api.h"
//
//using namespace cudacp;
//
//#ifndef MIN
//#define MIN(x,y) ((x < y) ? x : y)
//#endif
//
//static const int WORK_SIZE = 64;
//static const int U32_SIZE = sizeof(u32); ///<4
//static const int U32_BIT = U32_SIZE * 8;	///<32
//static const int U32_POS = 5;
//static const int U32_MOD_MASK = 31;
//
///// A single variable domain needs how many int, it can also be used in locate the index by variable's id;
//__constant__ int D_BITDOM_INTSIZE;
//__constant__ int D_NUM_BD_BLOCK;
//__constant__ int D_NUM_CS_SIZE_BLOCKS;
////////////////////////////////////////////////////////////////////////////
////	һЩGPU����
////////////////////////////////////////////////////////////////////////////
//__device__ __managed__ int NUM_BD_BLOCK;
//__device__ __managed__ int NUM_CS_SIZE_BLOCKS;
//// һ�����������int����
//__device__ __managed__ int BITDOM_INTSIZE;
//// �����������ϵ������int���� 
//__device__ __managed__ int BITDOMS_INTSIZE;
//// һ��Լ����bitsup��int���� 
//__device__ __managed__ int BITSUP_INTSIZE;
//// ����Լ�����ϵ�bitsup��int����
//__device__ __managed__ int BITSUPS_INTSIZE;
////�����������ܳ���
//__device__ __managed__ int BITSUBDOMS_INTSIZE;
//
////////////////////////////////////////////////////////////////////////////
// //GPUԼ����¼��Ϣ�����ɸ���
////////////////////////////////////////////////////////////////////////////
////	ÿ��������int��С
//__device__ __managed__ int *vars_size;
//// �洢Լ����scope������int3��scope.x: x.id; scope.y: y.id; scope.z: c.id
//__device__ __managed__ int3* scope;
//// ���dom
//__device__ __managed__ int MAX_DOM_SIZE;
//// subCon����
//__device__ __managed__ int SUBCON_SIZE;
//
//
////__device__ __managed__ int BITDOM_SIZE;
////__device__ __managed__ int
//// ���������ݽṹ��ʹ��UM
//// ��ʾԼ����������
//__device__ __managed__ u32* bitDom;
//// ��ʾԼ���������޸�
//__device__ __managed__ u32* bitSup;
////���ƶ��У��洢Լ��id
//__device__ __managed__ int *mainCon;
////���������ݽṹ
////��ʾ�������Լ����������
//__device__ __managed__ u32* bitSubDom;
////���ƶ��У��洢������Լ��id subCon.x: variable��subCon.y: value��subCon.z: c.id
//__device__ __managed__ ushort3* subCon;
////�����Ҫ��Լ��Լ��id����ʼȫ��Ϊ1
//__device__ __managed__ int* mainEvtCon;
////��Ƿ����Ķ��ı���id����ʼ��ȫ��Ϊ0
//__device__ __managed__ int* mainEvtVar;
//
//__device__ __managed__ ushort4* subVar;
////��������ⷢ���Ķ��ı���id����ʼ��ȫ��Ϊ0
//__device__ __managed__ unsigned short* subEvtVar;
////��������ⷢ���Ķ���Լ��id����ʼ��ȫ��Ϊ1
//__device__ __managed__ int* subEvtCon;
//
//static const u32 U32_MASK1[32] = {
//	0x80000000, 0x40000000, 0x20000000, 0x10000000,
//	0x08000000, 0x04000000, 0x02000000, 0x01000000,
//	0x00800000, 0x00400000, 0x00200000, 0x00100000,
//	0x00080000, 0x00040000, 0x00020000, 0x00010000,
//	0x00008000, 0x00004000, 0x00002000, 0x00001000,
//	0x00000800, 0x00000400, 0x00000200, 0x00000100,
//	0x00000080, 0x00000040, 0x00000020, 0x00000010,
//	0x00000008, 0x00000004, 0x00000002, 0x00000001,
//};
//
//static const u32 U32_MASK0[32] = {
//	0x7FFFFFFF, 0xBFFFFFFF, 0xDFFFFFFF, 0xEFFFFFFF,
//	0xF7FFFFFF, 0xFBFFFFFF, 0xFDFFFFFF, 0xFEFFFFFF,
//	0xFF7FFFFF, 0xFFBFFFFF, 0xFFDFFFFF, 0xFFEFFFFF,
//	0xFFF7FFFF, 0xFFFBFFFF, 0xFFFDFFFF, 0xFFFEFFFF,
//	0xFFFF7FFF, 0xFFFFBFFF, 0xFFFFDFFF, 0xFFFFEFFF,
//	0xFFFFF7FF, 0xFFFFFBFF, 0xFFFFFDFF, 0xFFFFFEFF,
//	0xFFFFFF7F, 0xFFFFFFBF, 0xFFFFFFDF, 0xFFFFFFEF,
//	0xFFFFFFF7, 0xFFFFFFFB, 0xFFFFFFFD, 0xFFFFFFFE,
//};
//
////__forceinline__ int  GetBitDomIndex(int var_id)
////{
////	return var_id * BITDOM_INTSIZE;
////}
//#define GetBitDomIndex(i,j) (i * BITDOM_INTSIZE + j)
//
//__inline__ __device__ __host__ int GetTopNum(int num_elements, int num_threads)
//{
//	return (num_elements + (num_threads - 1)) / num_threads;
//}
////************************************
//// Method:    intsizeof
//// FullName:  intsizeof
//// Access:    public 
//// Returns:   int
//// Qualifier: ��ȡ��bit��ʾ��int����
//// Parameter: const int x
////************************************
//inline int intsizeof(const int x)
//{
//	return (int)ceil((float)x / U32_BIT);
//}
//
//#define GetBitSupIndexByINTPrstn(cid,x_val,y_val) (cid * BITSUP_INTSIZE + x_val * MAX_DOM_SIZE + y_val)
//#define GetBitSupIndexByTuple(cid,t) (cid * BITSUP_INTSIZE + t.x  + t.y / U32_BIT * MAX_DOM_SIZE)
//void DelGPUModel();
//
//void BuildBitModel(XModel *xm)
//{
//#pragma region ���㳣��
//	BITDOM_INTSIZE = intsizeof(xm->feature.max_dom_size);
//	MAX_DOM_SIZE = xm->feature.max_dom_size;
//	BITDOMS_INTSIZE = BITDOM_INTSIZE * xm->feature.vs_size;
//	BITSUP_INTSIZE = xm->feature.max_dom_size * BITDOM_INTSIZE;
//	BITSUPS_INTSIZE = BITSUP_INTSIZE * xm->feature.cs_size;
//	BITSUBDOMS_INTSIZE = xm->feature.vs_size * xm->feature.max_dom_size * BITDOMS_INTSIZE;
//	SUBCON_SIZE = xm->feature.vs_size * xm->feature.ds_size * xm->feature.cs_size;
//#pragma endregion ���㳣��
//#pragma region Լ��������Ϣ
//	cudaMallocManaged(&vars_size, sizeof(int) * xm->feature.vs_size);
//	// ��ʼ���������С
//	for (int i = 0; i < xm->feature.vs_size; ++i)
//	{
//		XVar* v = xm->vars[i];
//		XDom* d = xm->doms[v->dom_id];
//		vars_size[i] = d->size;
//	}
//
//	// ��ʼ��scope
//	cudaMallocManaged(&scope, sizeof(int3) * xm->feature.cs_size);
//	for (int i = 0; i < xm->feature.cs_size; ++i)
//	{
//		XCon *c = xm->cons[i];
//		scope[i].x = c->scope[0];
//		scope[i].y = c->scope[1];
//		scope[i].z = c->id;
//	}
//#pragma endregion Լ��������Ϣ
//#pragma region ����bitDom
//	cudaMallocManaged(&bitDom, sizeof(int) * BITDOMS_INTSIZE);
//	cudaMallocManaged(&mainEvtVar, sizeof(int) * xm->feature.vs_size);
//
//	for (int i = 0; i < xm->feature.vs_size; ++i)
//	{
//		XVar* v = xm->vars[i];
//		XDom* d = xm->doms[v->dom_id];
//		int dom_size = d->size;
//		int dom_int_size = intsizeof(dom_size);
//
//		for (int j = 0; j < BITDOM_INTSIZE; ++j)
//		{
//			int idx = GetBitDomIndex(i, j);
//
//			if (j < dom_int_size - 1)
//			{
//				bitDom[idx] = UINT32_MAX;
//			}
//			else if (j == dom_int_size - 1)
//			{
//				int offset = U32_BIT - (dom_size & U32_MOD_MASK);
//				bitDom[idx] = UINT32_MAX << offset;
//			}
//			else
//			{
//				bitDom[idx] = 0;
//			}
//		}
//
//		mainEvtVar[i] = 1;
//	}
//
//	//for (int i = 0; i < xm->feature.vs_size; ++i)
//	//{
//	//	for (int j = 0; j < BITDOM_INTSIZE; ++j)
//	//	{
//	//		int idx = GetBitDomIndex(i, j);
//	//		printf("var = %d, j = %d, idx = %d, pre= %x\n", i, j, idx, bitDom[idx]);
//	//	}
//	//}
//#pragma endregion ����bitDom
//#pragma region ����bitSubDom
//
//#pragma endregion ����bitSubDom
//#pragma region ����bitCon
//	cudaMallocManaged(&bitSup, sizeof(int) * BITSUPS_INTSIZE);
//	for (int i = 0; i < xm->feature.cs_size; ++i)
//	{
//		XCon* c = xm->cons[i];
//		XRel* r = xm->rels[c->rel_id];
//		XVar* v[2] = { xm->vars[c->scope[0]],xm->vars[c->scope[1]] };
//		XDom* d[2] = { xm->doms[v[0]->dom_id], xm->doms[v[1]->dom_id] };
//
//		// ��ʼ��λ����
//		for (int j = 0; j < BITDOM_INTSIZE; ++j)
//		{
//			for (int k = 0; k < xm->feature.max_dom_size; ++k)
//			{
//				const int idx = GetBitSupIndexByINTPrstn(c->id, j, k);
//				if (k < d[0]->size && (j < (d[1]->size / U32_BIT)))
//				{
//					// ֧��ȡ0x0000...,��ͻȡ0xFFF...
//					bitSup[idx] = (r->sem == SEM_CONFLICT) ? UINT32_MAX : 0;
//				}
//				else if (j == (d[1]->size / U32_BIT))
//				{
//					bitSup[idx] = (r->sem == SEM_CONFLICT) ? UINT32_MAX : 0;
//					bitSup[idx] <<= U32_BIT - (d[1]->size & U32_MOD_MASK);
//				}
//				else
//				{
//					bitSup[idx] = 0;
//				}
//			}
//		}
//
//		// ��λ���������ֵ
//		for (int j = 0; j < r->size; ++j)
//		{
//			int2 t;
//			t.x = r->tuples[j][0];
//			t.y = r->tuples[j][1];
//
//			const int idx = GetBitSupIndexByTuple(c->id, t);
//
//			if (r->sem == SEM_SUPPORT)
//			{
//				bitSup[idx] |= U32_MASK1[t.y & U32_MOD_MASK];
//			}
//			else
//			{
//				bitSup[idx] &= U32_MASK0[t.y & U32_MOD_MASK];
//			}
//		}
//		//// ��ʼ��λ����
//		//for (int j = 0; j < BITDOM_INTSIZE; ++j)
//		//{
//		//	for (int k = 0; k < xm->feature.max_dom_size; ++k)
//		//	{
//		//		const int idx = GetBitSupIndexByINTPrstn(c->id, j, k);
//		//		printf("c_id = %d, j = %d, k = %d, idx = %d, pre= %x\n", i, j, k, idx, bitSup[idx]);
//		//	}
//		//}
//	}
//#pragma endregion ����bitCon
//#pragma region mainCon
//
//	cudaMallocManaged(&mainCon, sizeof(int)*xm->feature.cs_size);
//	cudaMallocManaged(&mainEvtCon, sizeof(int) * xm->feature.cs_size);
//
//	for (int i = 0; i < xm->feature.cs_size; ++i)
//	{
//		mainCon[i] = i;
//		mainEvtCon[i] = 0;
//	}
//
//#pragma endregion mainCon
//#pragma region subCon
//	cudaMallocManaged(&subCon, sizeof(int3)*SUBCON_SIZE);
//	for (int i = 0; i < xm->feature.vs_size; ++i)
//	{
//		XVar* v = xm->vars[i];
//		for (int j = 0; j < xm->feature.max_dom_size; ++j)
//		{
//			for (int k = 0; k < xm->feature.cs_size; ++k)
//			{
//				const int idx = i*xm->feature.max_dom_size*xm->feature.cs_size + j*xm->feature.cs_size + k;
//
//				subCon[idx].x = i;
//				subCon[idx].y = j;
//				subCon[idx].z = k;
//			}
//		}
//	}
//#pragma endregion subCon
//#pragma region ������������������
//#pragma endregion ������������������
//}
//
//void DelGPUModel()
//{
//	cudaFree(vars_size);
//	cudaFree(scope);
//	cudaFree(bitDom);
//	cudaFree(bitSup);
//}



